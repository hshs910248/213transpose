#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "transpose.h"
#include "tensor_util.h"
#include "cudacheck.h"

template<typename T>
void _213transpose(TensorUtil<T>& tu) {
	size_t& vol = tu.vol;
	//T* h_data = (T*)malloc(vol * sizeof(T));
	T* d_data = NULL;
	size_t dataSize = vol * sizeof(T);
	CudaSafeCall( hipMallocManaged(&d_data, dataSize) );
	tu.init_data(d_data);
	
	hipEvent_t start, stop;
	CudaSafeCall( hipEventCreate(&start) );
	CudaSafeCall( hipEventCreate(&stop) );
	CudaSafeCall( hipEventRecord(start, 0) );
	
	//CudaSafeCall( hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice) );
	
	int d1 = tu.dim[0];
	int d2 = tu.dim[1];
	int d3 = tu.dim[2];
	/*for (int k = 0; k < d3; k++) {
		inplace::transpose(true, d_data + k * d1 * d2, d2, d1);
	}*/
	inplace::transpose(d_data, d1, d2, d3);
	//CudaSafeCall( hipMemcpy(h_data, d_data, dataSize, hipMemcpyDeviceToHost) );
	CudaSafeCall( hipDeviceSynchronize() );
	CudaSafeCall( hipEventRecord(stop, 0) );
	CudaSafeCall( hipEventSynchronize(stop) );
	float t;
	CudaSafeCall( hipEventElapsedTime(&t, start, stop) );
	printf("Time: %.5fms\n", t);
	
	tu.write_file(d_data);
	
	CudaSafeCall( hipFree(d_data) );
}

int main(int argc, char** argv) {
	int dim[3];
	dim[0] = atoi(argv[1]);
	dim[1] = atoi(argv[2]);
	dim[2] = atoi(argv[3]);

	int type_size = atoi(argv[4]);
	FILE* fp = (argc == 6)? fopen(argv[5], "wb") : stdout;
	int permutation[3] = {1, 0, 2};
	if (type_size == 4) {
		TensorUtil<int> tu(fp, 3, dim, permutation);
		_213transpose<int>(tu);
	}
	else {
		TensorUtil<long long> tu(fp, 3, dim, permutation);
		_213transpose<long long>(tu);
	}
	if (fp != stdout) fclose(fp);
	
	return 0;
}
