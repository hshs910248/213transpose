#include <cstdio>
#include <cstdlib>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "transpose.h"
#include "tensor_util.h"
#include "cudacheck.h"
#include "skinny.h"

template<typename T>
void _213transpose(_213TensorUtil<T>& _213tu) {
	size_t vol = _213tu.vol;
	T* h_data = (T*)malloc(vol * sizeof(T));
	_213tu.init_data(h_data);
	
	_213tu.print_mat(h_data);
	
	hipEvent_t start, stop;
	CudaSafeCall( hipEventCreate(&start) );
	CudaSafeCall( hipEventCreate(&stop) );
	CudaSafeCall( hipEventRecord(start, 0) );
	
	T* d_data = NULL;
	size_t dataSize = vol * sizeof(T);
	CudaSafeCall( hipMalloc(&d_data, dataSize) );
	CudaSafeCall( hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice) );
	
	int d1 = _213tu.dim[0];
	int d2 = _213tu.dim[1];
	int d3 = _213tu.dim[2];
	for (int k = 0; k < d3; k++) {
		inplace::transpose(true, d_data + k * d1 * d2, d2, d1);
	}
	CudaSafeCall( hipMemcpy(h_data, d_data, dataSize, hipMemcpyDeviceToHost) );
	CudaSafeCall( hipDeviceSynchronize() );
	CudaSafeCall( hipEventRecord(stop, 0) );
	CudaSafeCall( hipEventSynchronize(stop) );
	float t;
	CudaSafeCall( hipEventElapsedTime(&t, start, stop) );
	//printf("Time: %.5fms\n", t);
	
	_213tu.print_mat(h_data);
	
	CudaSafeCall( hipFree(d_data) );
	free(h_data);
}

int main(int argc, char** argv) {
	int dim[3];
	dim[0] = atoi(argv[1]);
	dim[1] = atoi(argv[2]);
	dim[2] = atoi(argv[3]);

	int type_size = atoi(argv[4]);
	FILE* fp = (argc == 6)? fopen(argv[5], "wb") : stdout;
	size_t vol = (size_t)dim[0] * dim[1] * dim[2];
	if (type_size == 4) {
		_213TensorUtil<int> _213tu(fp, dim, vol, sizeof(dim));
		_213transpose<int>(_213tu);
	}
	else {
		_213TensorUtil<long long> _213tu(fp, dim, vol, sizeof(dim));
		_213transpose<long long>(_213tu);
	}
	
	return 0;
}