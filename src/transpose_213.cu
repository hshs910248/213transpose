#include "hip/hip_runtime.h"
#include <cstdio>
#include <algorithm>
#include "transpose.h"
#include "skinny_213.h"
#include "cudacheck.h"

namespace inplace {
namespace _213 {

template <typename T>
void transpose_fn(T* h_data, int d1, int d2, int d3) {
	//int numStream = d3;
	int numStream = std::min(d3, 512);
	
	hipStream_t* streams = new hipStream_t[numStream];
	for (int i = 0; i < numStream; i++) {
		hipStreamCreate(&streams[i]);
	}
	
	T* d_data = NULL;
	T** temp = new T*[numStream];
	size_t vol = (long long)d1 * d2 * d3;
	CudaSafeCall( hipMalloc(&d_data, vol * sizeof(T)) );
	for (int i = 0; i < numStream; i++) {
		CudaSafeCall( hipMalloc(&temp[i], std::max(d1, d2) * sizeof(T)) );
	}
	size_t chunkSize = (long long)d1 * d2 * sizeof(T);
	for (int k = 0; k < d3; k++) {
		long long offset = (long long)k * d1 * d2;
		int sid = k % numStream;
		CudaSafeCall( hipMemcpyAsync(d_data + offset, h_data + offset, chunkSize, hipMemcpyHostToDevice, streams[sid]) );
		_2d::transpose(streams[sid], true, (float*)(d_data + offset), (float*)temp[sid], d2, d1);
		CudaSafeCall( hipMemcpyAsync(h_data + offset, d_data + offset, chunkSize, hipMemcpyDeviceToHost, streams[sid]) );
	}
	CudaSafeCall( hipDeviceSynchronize() );
	
	CudaSafeCall( hipFree(d_data) );
	for (int i = 0; i < numStream; i++) {
		CudaSafeCall( hipFree(temp[i]) );
		CudaSafeCall( hipStreamDestroy(streams[i]) );
	}
}

template <typename T>
void skinny_transpose(T* h_data, int d1, int d2, int d3, bool small_m, bool small_n) {
	size_t value;
	CudaSafeCall( hipDeviceGetLimit(&value, hipLimitStackSize) );
    CudaSafeCall( hipDeviceSetLimit(hipLimitStackSize, value * 32) );
	
	T* d_data = NULL;
	size_t vol = (long long)d1 * d2 * d3;
	CudaSafeCall( hipMalloc(&d_data, vol * sizeof(T)) );
	
	int m = d2;
	int n = d1;
	if (!small_m && small_n) {
        std::swap(m, n);
    }
	int sid = 0;
	int numStream = (small_n)? std::min(d3, 128) : min(d3, m);
	hipStream_t* streams = new hipStream_t[numStream];
	T** temp = new T*[numStream];
	for (int i = 0; i < numStream; i++) {
		hipStreamCreate(&streams[i]);
		if (!small_n) {
			CudaSafeCall( hipMalloc(&temp[i], sizeof(T) * n * m) );
		}
		else {
			temp[i] = NULL;
		}
	}
	int h = (small_n)? d3 / (1024 * 2) : min(d3, m);
	for (int k = 0; k < d3; k += h) {
		long long offset = (long long)k * d1 * d2;
		if (h < d3 && d3 - k < h) {
			h = d3 - k;
		}
		size_t chunkSize = (long long)h * d1 * d2 * sizeof(T);
		CudaSafeCall( hipMemcpyAsync(d_data + offset, h_data + offset, chunkSize, hipMemcpyHostToDevice, streams[sid]) );
		if (!small_m && small_n) {
			r2c::skinny_transpose(streams[sid], d_data + offset, temp[sid], n, m, h);
		}
		else {
			c2r::skinny_transpose(streams[sid], d_data + offset, temp[sid], n, m, h);
		}
		CudaSafeCall( hipMemcpyAsync(h_data + offset, d_data + offset, chunkSize, hipMemcpyDeviceToHost, streams[sid]) );
		sid = (sid + 1) % numStream;
	}
	CudaSafeCall( hipDeviceSynchronize() );
	CudaSafeCall( hipFree(d_data) );
	for (int i = 0; i < numStream; i++) {
		CudaSafeCall( hipStreamDestroy(streams[i]) );
	}
}

template <typename T>
void transpose(T* data, int d1, int d2, int d3) {
	bool small_m = (d2 <= 32);
    bool small_n = (d1 <= 32);
	
	hipEvent_t start, stop;
	CudaSafeCall( hipEventCreate(&start) );
	CudaSafeCall( hipEventCreate(&stop) );
	CudaSafeCall( hipEventRecord(start, 0) );
	
	if (small_m || small_n) {
		//printf("skinny_transpose\n");
		skinny_transpose(data, d1, d2, d3, small_m, small_n);
	}
	else {
		//printf("transpose_fn\n");
		transpose_fn(data, d1, d2, d3);
	}
	
	CudaSafeCall( hipEventRecord(stop, 0) );
	CudaSafeCall( hipEventSynchronize(stop) );
	float t;
	CudaSafeCall( hipEventElapsedTime(&t, start, stop) );
	printf("Time: %.5fms\n", t);
}

template void transpose(int*, int, int, int);
template void transpose(long long*, int, int, int);
template void transpose(float*, int, int, int);
template void transpose(double*, int, int, int);

}
}