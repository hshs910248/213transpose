#include "hip/hip_runtime.h"
#include <set>
#include <vector>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/transform.h>
#include "gcd.h"
#include "index.h"
#include "introspect.h"
#include "util.h"
#include "equations.h"
#include <cstdio>

namespace inplace {
namespace detail {
    
template<typename Fn>
void scatter_cycles(Fn f, std::vector<int>& heads, std::vector<int>& lens) {
    int len = f.len();
    //printf("len = %d\n", len);
    thrust::counting_iterator<int> i(0);
    std::set<int> unvisited(i, i+len);
    while(!unvisited.empty()) {
        int idx = *unvisited.begin();
        //printf("idx = %d\n", idx);
        unvisited.erase(unvisited.begin());
        int dest = f(idx);
        if (idx != dest) {
            heads.push_back(idx);
            int start = idx;
            int len = 1;
            //std::cout << "Cycle: " << start << " " << dest << " ";
            while(dest != start) {
                idx = dest;
                unvisited.erase(idx);
                dest = f(idx);
                //printf("f(%d) = %d\n", idx, dest);
                len++;
                //std::cout << dest << " ";
            }
            //std::cout << std::endl;
            lens.push_back(len);
        }
    }
}


template<typename T, typename F, int U>
__device__ __forceinline__ void unroll_cycle_row_permute(
    F f, row_major_index rm, T* data, int i, int j, int l) {
    
    T src = data[rm(i, j)];
    T loaded[U+1];
    loaded[0] = src;
    for(int k = 0; k < l / U; k++) {
        int rows[U];
#pragma unroll
        for(int x = 0; x < U; x++) {
            i = f(i);
            rows[x] = i;
        }
#pragma unroll
        for(int x = 0; x < U; x++) {
            loaded[x+1] = data[rm(rows[x], j)];
        }
#pragma unroll
        for(int x = 0; x < U; x++) {
            data[rm(rows[x], j)] = loaded[x];
        }
        loaded[0] = loaded[U];
    }
    T tmp = loaded[0];
    for(int k = 0; k < l % U; k++) {
        i = f(i);
        T new_tmp = data[rm(i, j)];
        data[rm(i, j)] = tmp;
        tmp = new_tmp;
    }
    
}

template<typename T, typename F, int U>
__global__ void cycle_row_permute(F f, T* data, int* heads,
                                  int* lens, int n_heads) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int d1 = f.n;
    row_major_index rm(f.m, f.n);


    if ((j < d1) && (h < n_heads)) {
        int i = heads[h];
        int l = lens[h];
        unroll_cycle_row_permute<T, F, U>(f, rm, data, i, j, l);
    }
}

template<typename T, typename F>
void scatter_permute(F f, int d3, int d2, int d1, T* data, int* tmp) {
    std::vector<int> heads;
    std::vector<int> lens;
    scatter_cycles(f, heads, lens);
    int* d_heads = tmp;
    int* d_lens = tmp + d2 / 2;
    hipMemcpy(d_heads, heads.data(), sizeof(int)*heads.size(),
               hipMemcpyHostToDevice);
    hipMemcpy(d_lens, lens.data(), sizeof(int)*lens.size(),
               hipMemcpyHostToDevice);
    int n_threads_x = 256;
    int n_threads_y = 1024/n_threads_x;
    
    int n_blocks_x = div_up(d1, n_threads_x);
    int n_blocks_y = div_up(heads.size(), n_threads_y);
    
    size_t d1d2 = (size_t)d1 * (size_t)d2;
	for (size_t i = 0; i < d3; i++) {
        size_t offset = i * d1d2;
        cycle_row_permute<T, F, 4>
        <<<dim3(n_blocks_x, n_blocks_y),
        dim3(n_threads_x, n_threads_y)>>>
        (f, data + offset, d_heads, d_lens, heads.size());
    }
}


template void scatter_permute(c2r::scatter_postpermuter, int, int, int, float*, int*);
template void scatter_permute(c2r::scatter_postpermuter, int, int, int, double*, int*);
template void scatter_permute(c2r::scatter_postpermuter, int, int, int, int*, int*);
template void scatter_permute(c2r::scatter_postpermuter, int, int, int, long long*, int*);

template void scatter_permute(r2c::scatter_prepermuter, int, int, int, float*, int*);
template void scatter_permute(r2c::scatter_prepermuter, int, int, int, double*, int*);
template void scatter_permute(r2c::scatter_prepermuter, int, int, int, int*, int*);
template void scatter_permute(r2c::scatter_prepermuter, int, int, int, long long*, int*);


}
}
