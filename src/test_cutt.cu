#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <algorithm>
#include <hip/hip_runtime.h>
#include "cutt.h"
#include "tensor_util.h"
#include "cudacheck.h"

#define cuttCheck(stmt) do {                                 \
  cuttResult err = stmt;                            \
  if (err != CUTT_SUCCESS) {                          \
    fprintf(stderr, "%s in file %s, function %s\n", #stmt,__FILE__,__FUNCTION__); \
    exit(1); \
  }                                                  \
} while(0)

template<typename T>
float cutt_plan(T* idata, T* odata, int* dim, int* permutation, size_t dataSize) {
	hipEvent_t start, stop;
	CudaSafeCall( hipEventCreate(&start) );
	CudaSafeCall( hipEventCreate(&stop) );
	CudaSafeCall( hipEventRecord(start, 0) );
	
	cuttHandle plan;
	cuttCheck(cuttPlan(&plan, 3, dim, permutation, sizeof(T), 0));
    
    //int dev;
    //CudaSafeCall( hipGetDevice(&dev) );
    //CudaSafeCall( hipMemPrefetchAsync(idata, dataSize, dev, 0) );
	cuttCheck(cuttExecute(plan, idata, odata));
	
	CudaSafeCall( hipDeviceSynchronize() );
	CudaSafeCall( hipEventRecord(stop, 0) );
	CudaSafeCall( hipEventSynchronize(stop) );
	float t;
	CudaSafeCall( hipEventElapsedTime(&t, start, stop) );

	cuttCheck(cuttDestroy(plan));
	return t;
}

template<typename T>
float cutt_plan_measure(T* idata, T* odata, int* dim, int* permutation, size_t dataSize) {
	hipEvent_t start, stop;
	CudaSafeCall( hipEventCreate(&start) );
	CudaSafeCall( hipEventCreate(&stop) );
	CudaSafeCall( hipEventRecord(start, 0) );
	
	cuttHandle plan;
	cuttPlanMeasure(&plan, 3, dim, permutation, sizeof(T), 0, idata, odata);
    //int dev;
    //CudaSafeCall( hipGetDevice(&dev) );
    //CudaSafeCall( hipMemPrefetchAsync(idata, dataSize, dev, 0) );
	cuttCheck(cuttExecute(plan, idata, odata));
	
	CudaSafeCall( hipDeviceSynchronize() );
	CudaSafeCall( hipEventRecord(stop, 0) );
	CudaSafeCall( hipEventSynchronize(stop) );
	float t;
	CudaSafeCall( hipEventElapsedTime(&t, start, stop) );

	cuttCheck(cuttDestroy(plan));
	return t;
}

template<typename T>
void test_cutt(TensorUtil<T>& tu) {
	size_t& vol = tu.vol;
	T* i_data = NULL;
	size_t dataSize = vol * sizeof(T);
	CudaSafeCall( hipMallocManaged(&i_data, dataSize) );
	tu.init_data(i_data);
	
	int dim[3] = {(int)tu.dim[0], (int)tu.dim[1], (int)tu.dim[2]};
	int permutation[3] = {1, 0, 2};
	
	//T* ans = (T*)malloc(dataSize);
	//tu.seq_tt(ans, i_data);
    /*T* seq_i_data = (T*)malloc(dataSize);
	memcpy(seq_i_data, i_data, dataSize);
	T* seq_o_data = (T*)malloc(dataSize);
    tu.seq_tt(seq_o_data, seq_i_data);*/

	T* o_data = NULL;
	CudaSafeCall( hipMallocManaged(&o_data, dataSize) );
	float t1 = cutt_plan(i_data, o_data, dim, permutation, dataSize);
	
	CudaSafeCall( hipFree(i_data) );
	CudaSafeCall( hipFree(o_data) );
	
	//CudaSafeCall( hipMallocManaged(&i_data, dataSize) );
	//tu.init_data(i_data);
	//CudaSafeCall( hipMallocManaged(&o_data, dataSize) );
	
	//float t2 = cutt_plan_measure(i_data, o_data, dim, permutation, dataSize);

	printf("Time: %.5fms\n", t1);
    FILE* txtfp = fopen("cutt_time.txt", "a+");
    fprintf(txtfp, "%.5f\n", t1);
    fclose(txtfp);
    //printf("%.5f\n", std::min(t1, t2));
    //if (memcmp(seq_o_data, o_data, dataSize)) printf("Error\n");
	
	//CudaSafeCall( hipFree(i_data) );
	//CudaSafeCall( hipFree(o_data) );
	//free(ans);
}

int main(int argc, char** argv) {
	int dim[3];
	dim[0] = atoi(argv[1]);
	dim[1] = atoi(argv[2]);
	dim[2] = atoi(argv[3]);
	int permutation[3] = {1, 0, 2};
	//printf("Data Size = %lld bytes\n", (long long)dataSize);
	
	int type_size = atoi(argv[4]);
	FILE* fp = (argc == 6)? fopen(argv[5], "wb") : stdout;
	if (type_size == 4) {
		TensorUtil<int> tu(fp, 3, dim, permutation);
		test_cutt<int>(tu);
	}
	else {
		TensorUtil<long long> tu(fp, 3, dim, permutation);
		test_cutt<long long>(tu);
	}
	if (fp != stdout) fclose(fp);
}
